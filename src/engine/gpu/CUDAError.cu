//
// Created by chpon on 26/09/2018.
//

#include "CUDAError.h"
#include <hip/hip_runtime.h>

CUDAError::CUDAError() : Engine::Error(
        std::string(hipGetErrorName(hipGetLastError())) + ": " + hipGetErrorString(hipGetLastError())) {
}

CUDAError::CUDAError(const std::string &message) : Engine::Error(message) {
}
